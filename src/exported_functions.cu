#include "hip/hip_runtime.h"
#include "neural-graphics-primitives/exported_functions.h"

#ifdef _WIN32
#include <GL/gl3w.h>
#else
#include <GL/glew.h>
#endif
#include <GLFW/glfw3.h>
#include "gl/GL.h"
#include "gl/GLU.h"

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/random_val.cuh>
#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/camera_path.h>
#include <neural-graphics-primitives/discrete_distribution.h>
#include <neural-graphics-primitives/nerf.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/sdf.h>
#include <neural-graphics-primitives/shared_queue.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>
#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/gpu_memory.h>
#include <filesystem/path.h>
#include <cuda_gl_interop.h>

#include <tiny-cuda-nn/multi_stream.h>
#include <tiny-cuda-nn/random.h>

#include <json/json.hpp>
#include <filesystem/path.h>
#include <thread>
#include "gl/GL.h"
#include "gl/GLU.h"
#include <memory>

using Texture = std::shared_ptr<ngp::GLTexture>;
using RenderBuffer = std::shared_ptr<ngp::CudaRenderBuffer>;
using namespace Eigen;

// FIXME por qué hay que redefinirlas?
inline constexpr float PI() { return 3.14159265358979323846f; }
inline NGP_HOST_DEVICE Eigen::Vector2f fov_to_focal_length(const Eigen::Vector2i &resolution, const Eigen::Vector2f &degrees)
{
	return 0.5f * resolution.cast<float>().cwiseQuotient((0.5f * degrees * (float)PI() / 180).array().tan().matrix());
}

struct TextureData
{
	TextureData(const Texture &tex, const RenderBuffer &buf, int width, int heigth)
		: surface_texture(tex), render_buffer(buf), width(width), height(height)
	{
	}

	Texture surface_texture;
	RenderBuffer render_buffer;
	int width;
	int height;
};

static bool already_initalized = false;
static std::shared_ptr<ngp::Testbed> testbed = nullptr;

/*
Check Testbed::init_window and main::main_func
*/
extern "C" void nerf_initialize(const char *scene, const char *snapshot, bool dlss)
{
	if (already_initalized)
	{
		std::cout << "Already initalized nerf" << std::endl;
		return;
	}

	testbed = std::make_shared<ngp::Testbed>(
		ngp::ETestbedMode::Nerf,
		scene);

	if (snapshot)
	{
		testbed->load_snapshot(
			snapshot);
	}

	if (!glfwInit())
	{
		std::cout << "Could not initialize glfw" << std::endl;
	}
	if (gl3wInit() != 0)
	{
		std::cout << "Could not initialize gl3w" << std::endl;
	}

#ifdef NGP_VULKAN
	if (dlss)
	{
		try
		{
			testbed->m_dlss_provider = ngp::init_vulkan_and_ngx();
			if (testbed->m_testbed_mode == ngp::ETestbedMode::Nerf)
			{
				testbed->m_aperture_size = 0.0f;
				testbed->m_dlss = true;
			}
		}
		catch (const std::runtime_error &e)
		{
			tlog::warning() << "Could not initialize Vulkan and NGX. DLSS not supported. (" << e.what() << ")";
		}
	}
#endif

	already_initalized = true;
}

extern "C" void nerf_deinitialize()
{
	testbed->m_views.clear();
	testbed->m_rgba_render_textures.clear();
	testbed->m_depth_render_textures.clear();

	testbed->m_pip_render_buffer.reset();
	testbed->m_pip_render_texture.reset();

	testbed->m_dlss = false;
	testbed->m_dlss_provider.reset();
	testbed.reset();
	glfwTerminate();

	already_initalized = false;
}

/*
Check Testbed::begin_vr_frame_and_handle_vr_input, Testbed::init_window
*/
extern "C" void nerf_create_textures(int num_views, float *fov, int width, int height, float scaleFactor, unsigned int *handles)
{
	if (!testbed)
		return;

	testbed->set_n_views(num_views);
	testbed->m_foveated_rendering = false; // TODO foveated rendering

	// set fov and screen center
	if (num_views == 1)
	{
		// Desktop render
		testbed->m_views[0].relative_focal_length = testbed->m_relative_focal_length;
		testbed->m_views[0].screen_center = testbed->m_screen_center;
	}
	else
	{
		// VR render
		for (int i = 0; i < num_views; i++)
		{
			float angleLeft = fov[i * 4];
			float angleRight = fov[(i * 4) + 1];
			float angleUp = fov[(i * 4) + 2];
			float angleDown = fov[(i * 4) + 3];

			// Compute the distance on the image plane (1 unit away from the camera) that an angle of the respective FOV spans
			Vector2f rel_focal_length_left_down = 0.5f * fov_to_focal_length(Vector2i::Ones(), Vector2f{360.0f * angleLeft / PI(), 360.0f * angleDown / PI()});
			Vector2f rel_focal_length_right_up = 0.5f * fov_to_focal_length(Vector2i::Ones(), Vector2f{360.0f * angleRight / PI(), 360.0f * angleUp / PI()});
			testbed->m_views[i].relative_focal_length = rel_focal_length_right_up - rel_focal_length_left_down;

			// Compute fraction of that distance that is spanned by the right-up part and set screen center accordingly.
			Vector2f ratio = rel_focal_length_right_up.cwiseQuotient(testbed->m_views[i].relative_focal_length);
			testbed->m_views[i].screen_center = {1.0f - ratio.x(), ratio.y()};
		}
	}

	// create textures and dlss
	for (int i = 0; i < num_views; i++)
	{
		// TODO render on different GPUs if available
		testbed->m_views[i].device = &(testbed->primary_device()); // Render each view on primary GPU
		testbed->m_views[i].full_resolution = {width, height};
		testbed->m_views[i].render_buffer->set_hidden_area_mask(nullptr);
		testbed->m_views[i].visualized_dimension = -1;
		testbed->m_views[i].foveation = {}; // TODO foveated rendering

		// dlss with scaled resolution
		auto buffer = testbed->m_views[i].render_buffer;
		auto full_resolution = testbed->m_views[i].full_resolution;

		if (testbed->m_dlss)
		{
			buffer->enable_dlss(*testbed->m_dlss_provider, full_resolution);
		}
		else
		{
			buffer->disable_dlss();
		}

		Eigen::Vector2i render_res = buffer->in_resolution();
		Eigen::Vector2i new_render_res = (full_resolution.cast<float>() * scaleFactor).cast<int>().cwiseMin(full_resolution).cwiseMax(full_resolution / 16);

		float ratio = std::sqrt((float)render_res.prod() / (float)new_render_res.prod());
		if (ratio > 1.2f || ratio < 0.8f || scaleFactor == 1.0f || !testbed->m_dynamic_res)
		{
			render_res = new_render_res;
		}

		if (buffer->dlss())
		{
			render_res = buffer->dlss()->clamp_resolution(render_res);
			buffer->dlss()->update_feature(render_res, buffer->dlss()->is_hdr(), buffer->dlss()->sharpen());
		}

		buffer->resize(render_res);

		auto texture = testbed->m_rgba_render_textures[i];
		handles[i] = texture->texture();
	}
}

extern "C" void nerf_update_textures(float *camera_matrix)
{
	if (!testbed)
		return;

	int num_views = testbed->m_views.size();
	if (num_views > 1)
	{
		testbed->reset_accumulation(true);
	}
	else
	{
		testbed->reset_accumulation();
	}

	if (testbed->m_dlss)
	{
		testbed->m_aperture_size = 0.0f;
		if (!ngp::supports_dlss(testbed->m_nerf.render_lens.mode))
		{
			testbed->m_nerf.render_with_lens_distortion = false;
		}
	}

	// TODO update dynamic res and DLSS #2706

	// TODO foveated rendering #2758

	for (int i = 0; i < num_views; i++)
	{
		auto matrix_slice = &camera_matrix[i * 12];
		Eigen::Matrix<float, 3, 4> camera{matrix_slice};

		auto &view = testbed->m_views[i];
		testbed->render_frame(testbed->m_stream.get(),
							  camera,
							  camera,
							  camera,
							  view.screen_center,
							  view.relative_focal_length,
							  {0.0f, 0.0f, 0.0f, 1.0f},
							  view.foveation,
							  view.prev_foveation,
							  view.visualized_dimension,
							  *view.render_buffer,
							  true,
							  view.device);
	}
}

// utility functions

extern "C" void nerf_set_fov(float val)
{
	if (!testbed || testbed->m_views.size() != 1)
		return;

	testbed->set_fov(val);
	testbed->m_views[0].relative_focal_length = testbed->m_relative_focal_length;
}

extern "C" void nerf_update_aabb_crop(float *min_vec, float *max_vec)
{
	if (!testbed)
		return;

	Eigen::Vector3f min_aabb{min_vec};
	Eigen::Vector3f max_aabb{max_vec};

	testbed->m_render_aabb = ngp::BoundingBox(min_aabb, max_aabb);
}

extern "C" void nerf_reset_camera()
{
	if (!testbed)
		return;
	testbed->reset_camera();
}
