#include "hip/hip_runtime.h"
#include "neural-graphics-primitives/exported_functions.h"

#ifdef _WIN32
#include <GL/gl3w.h>
#else
#include <GL/glew.h>
#endif
#include <GLFW/glfw3.h>
#include "gl/GL.h"
#include "gl/GLU.h"

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/random_val.cuh>
#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/camera_path.h>
#include <neural-graphics-primitives/discrete_distribution.h>
#include <neural-graphics-primitives/nerf.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/sdf.h>
#include <neural-graphics-primitives/shared_queue.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>
#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/gpu_memory.h>
#include <filesystem/path.h>
#include <cuda_gl_interop.h>

#include <tiny-cuda-nn/multi_stream.h>
#include <tiny-cuda-nn/random.h>

#include <json/json.hpp>
#include <filesystem/path.h>
#include <thread>
#include "gl/GL.h"
#include "gl/GLU.h"
#include <memory>

using Texture = std::shared_ptr<ngp::GLTexture>;
using RenderBuffer = std::shared_ptr<ngp::CudaRenderBuffer>;
using namespace Eigen;

// FIXME por qué hay que redefinirlas?
inline constexpr float PI() { return 3.14159265358979323846f; }
inline NGP_HOST_DEVICE Eigen::Vector2f fov_to_focal_length(const Eigen::Vector2i &resolution, const Eigen::Vector2f &degrees)
{
	return 0.5f * resolution.cast<float>().cwiseQuotient((0.5f * degrees * (float)PI() / 180).array().tan().matrix());
}

struct TextureData
{
	TextureData(const Texture &tex, const RenderBuffer &buf, int width, int heigth)
		: surface_texture(tex), render_buffer(buf), width(width), height(height)
	{
	}

	Texture surface_texture;
	RenderBuffer render_buffer;
	int width;
	int height;
};

static bool already_initalized = false;
static bool use_dlss = false;
static uint32_t nullHandle;
static std::shared_ptr<ngp::Testbed> testbed = nullptr;
static std::unordered_map<GLuint, std::shared_ptr<TextureData>> textures;

extern "C" void nerf_initialize(const char *scene, const char *snapshot, bool dlss)
{
	if (already_initalized)
	{
		std::cout << "Already initalized nerf" << std::endl;
		return;
	}

	use_dlss = dlss;
	already_initalized = true;

	testbed = std::make_shared<ngp::Testbed>(
		ngp::ETestbedMode::Nerf,
		scene);

	if (snapshot)
	{
		testbed->load_snapshot(
			snapshot);
	}

	if (!glfwInit())
	{
		std::cout << "Could not initialize glfw" << std::endl;
	}
	if (!gl3wInit())
	{
		std::cout << "Could not initialize gl3w" << std::endl;
	}

#ifdef NGP_VULKAN
	if (use_dlss)
	{
		try
		{
			ngp::vulkan_and_ngx_init();
		}
		catch (std::runtime_error exception)
		{
			std::cout << "Could not initialize vulkan" << std::endl;
		}
	}
#endif
}

extern "C" void nerf_deinitialize()
{

#ifdef NGP_VULKAN
	if (use_dlss)
	{
		ngp::vulkan_and_ngx_destroy();
	}
#endif
	already_initalized = false;
	testbed.reset();
	glfwTerminate();
}

extern "C" unsigned int nerf_create_texture(int width, int height)
{
	if (!testbed)
		return 0;

	// gladly ngp already implements gl textures for us
	// so we just need to call GLTexture to create a new one.
	auto texture = std::make_shared<ngp::GLTexture>();
	auto buffer = std::make_shared<ngp::CudaRenderBuffer>(texture);

	Eigen::Vector2i render_res{width, height};
#if defined(NGP_VULKAN)
	if (use_dlss)
	{
		buffer->enable_dlss({width, height});
		// buffer->resize({ width, height });

		Eigen::Vector2i texture_res{width, height};
		render_res = buffer->in_resolution();
		if (render_res.isZero())
		{
			render_res = texture_res / 16;
		}
		else
		{
			render_res = render_res.cwiseMin(texture_res);
		}

		if (buffer->dlss())
		{
			render_res = buffer->dlss()->clamp_resolution(render_res);
		}
	}
#endif

	buffer->resize(render_res);

	GLuint handle = texture->texture();

	textures[texture->texture()] = std::make_shared<TextureData>(
		texture,
		buffer,
		width,
		height);

	return handle;
}

extern "C" void nerf_set_fov(float val)
{
	if (!testbed)
		return;

	testbed->set_fov(val);
}

extern "C" void nerf_update_texture(float *camera_matrix, unsigned int handle, float *fov)
{
	if (!testbed)
		return;

	auto found = textures.find(handle);
	if (found == std::end(textures))
	{
		return;
	}

	Eigen::Matrix<float, 3, 4> camera{camera_matrix};

	/*********Set fov for view*********/
	float angleLeft = fov[0];
	float angleRight = fov[1];
	float angleUp = fov[2];
	float angleDown = fov[3];

	// Compute the distance on the image plane (1 unit away from the camera) that an angle of the respective FOV spans
	Vector2f rel_focal_length_left_down = 0.5f * fov_to_focal_length(Vector2i::Ones(), Vector2f{360.0f * angleLeft / PI(), 360.0f * angleDown / PI()});
	Vector2f rel_focal_length_right_up = 0.5f * fov_to_focal_length(Vector2i::Ones(), Vector2f{360.0f * angleRight / PI(), 360.0f * angleUp / PI()});

	// Compute total distance (for X and Y) that is spanned on the image plane.
	testbed->m_relative_focal_length = rel_focal_length_right_up - rel_focal_length_left_down;

	// Compute fraction of that distance that is spanned by the right-up part and set screen center accordingly.
	Vector2f ratio = rel_focal_length_right_up.cwiseQuotient(testbed->m_relative_focal_length);
	testbed->m_screen_center = {1.0f - ratio.x(), ratio.y()};

	// Fix up weirdness in the rendering pipeline
	// // relative_focal_length[(m_fov_axis + 1) % 2] *= (float)view_resolution[(m_fov_axis + 1) % 2] / (float)view_resolution[m_fov_axis];

	RenderBuffer render_buffer = found->second->render_buffer;
	render_buffer->reset_accumulation();
	testbed->render_frame(camera,
						  camera,
						  Eigen::Vector4f::Zero(),
						  *render_buffer.get(),
						  true);
}

extern "C" void nerf_update_aabb_crop(float *min_vec, float *max_vec)
{
	if (!testbed)
		return;

	Eigen::Vector3f min_aabb{min_vec};
	Eigen::Vector3f max_aabb{max_vec};

	testbed->m_render_aabb = ngp::BoundingBox(min_aabb, max_aabb);
}

extern "C" void nerf_destroy_texture(unsigned int handle)
{
	if (!testbed)
		return;

	// @TODO add warnings and stuff
	// GLuint handle = static_cast<GLuint>(*handle_ptr);
	auto found = textures.find(handle);
	if (found == std::end(textures))
	{
		return;
	}

	found->second->surface_texture.reset();
	found->second->render_buffer.reset();

	found->second.reset();
}

// utility functions

extern "C" void nerf_reset_camera()
{
	if (!testbed)
		return;
	testbed->reset_camera();
}
